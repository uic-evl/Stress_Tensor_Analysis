#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include "reductionMax.hh"
#include "volumeRender_kernel.cuh"

typedef unsigned int  uint;
typedef unsigned char uchar;

typedef struct {
    float4 m[3];
} float3x4;

typedef unsigned short VolumeType;
//typedef float VolumeType;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix
texture<VolumeType, 3, hipReadModeElementType> tex;         // 3D texture
texture<VolumeType, 3, hipReadModeElementType> tex_cluster;         // 3D texture

struct Ray {
	float3 o;	// origin
	float3 d;	// direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__
float4 color_interpolate_cluster(float sample){
	
	// ACCENT
	
	// if(sample <= 1)
	// 	return make_float4((float)0.99215,(float)0.75294, (float)0.52549, 1.0);
	// else if(sample <= 2)
	// 	return make_float4( (float)0.498, (float)0.7882, (float)0.498, 0.25);
	// else if(sample <= 3)
	// 	return make_float4((float)0.74509,(float)0.68235, (float)0.83137, 1.0);
	// else if(sample <= 4)
	// 	return make_float4(1.0,1.0,1.0,1.0);
	
	// Dark2
	
	if(sample <= 1)
		return make_float4( 0.8509803921569,0.3725490196078,0.007843137254902, 1.0);
	else if(sample <= 2)
		return make_float4( 0.1058823529412, 0.6196078431373, 0.4666666666667, 0.25);
	else if(sample <= 3)
		return make_float4( 0.4588235294118,0.4392156862745,0.7019607843137, 1.0);
	else if(sample <= 4)
		return make_float4(1.0,1.0,1.0,1.0);
	
	return make_float4(0.0,0.0,0.0,0.0);	
}
__device__ 
float4 color_interpolate_large(float sample, float4 one, float4 two, float4 three,
				float4 four, float4 five, float4 six){
	
	float4 retcolor = make_float4(0);
	float percent = 0.0f; 
		
	if(sample <= 0.2f){
	
		percent = (0.2f - sample) / 0.2f;
		retcolor = (percent)*one + (1.0f-percent) * two;
		
	}else if(sample > 0.2f && sample <= 0.3f){
		
		percent = (0.3f - sample)  / 0.1f;
		retcolor = (percent)*two + (1.0f-percent) * three;
		
	}else if(sample > 0.3f && sample <= 0.4f){
		
		percent = (0.4f - sample) / 0.1f;
		retcolor = (percent)*three + (1.0f-percent) * four;
		
	}else if(sample > 0.4f && sample <= 0.5f){
		
		percent = (0.5f - sample) / 0.1f;
		retcolor = (percent)*four + (1.0f-percent) * five;
		
	}else{
		
		percent = (1.0 - sample) / 0.5f;
		retcolor = (percent)*five + (1.0f-percent) * six;
	}
	
	return retcolor;	
}
__device__ 
float4 color_interpolate(float sample, float4 one, float4 two, float4 three,
				float4 four, float4 five, float4 six){
	
	float4 retcolor = make_float4(0);
	float percent = 0.0f; 
		
	if(sample <= 25500.0f){
	
		percent = (25500.0f - sample) / 25500.0f;
		retcolor = (percent)*one + (1.0f-percent) * two;
		
	}else if(sample > 25500.0f && sample <= 26500.0f){
		
		percent = (26500.0f - sample)  / 1000.0f;
		retcolor = (percent)*two + (1.0f-percent) * three;
		
	}else if(sample > 26500.0f && sample <= 27500.0f){
		
		percent = (27500.0f - sample) / 1000.0f;
		retcolor = (percent)*three + (1.0f-percent) * four;
		
	}else if(sample > 27500.0f && sample <= 28500.0f){
		
		percent = (28500.0f - sample) / 1000.0f;
		retcolor = (percent)*four + (1.0f-percent) * five;
		
	}else{
		
		percent = (65535.0f - sample) / 65535.0f;
		retcolor = (percent)*five + (1.0f-percent) * six;
	}
	
	return retcolor;	
}

__device__ uint rgbaFloatToInt(float4 rgba, float global_max, float red, float green, float blue)
{
	rgba.x = rgba.x / (global_max+2);
    rgba.y = rgba.y / (global_max+2);
    rgba.z = rgba.z / (global_max+2);
	rgba.w = 0.5;
	
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__ void
d_render(float4 *d_iColors, ushort *data,
 						float *d_iRed, float *d_iGreen, float *d_iBlue, uint imageW, uint imageH,
     					float density, float brightness, float4 one, float4 two, float4 three, 
						float4 four, float4 five, float4 six, int type)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;
    float sample = 0;

    for(int i=0; i<maxSteps; i++) {
        
		// read from 3D texture
       // remap position to [0, 1] coordinates
  		if(type == 0)
			sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
		else
			sample = tex3D(tex_cluster, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
			float4 col = make_float4(0.0f);

        // lookup in transfer function texture
		if(type == 0)
			col = color_interpolate(sample,one,two,three,four,five,six);
		else
			col = color_interpolate_cluster(sample);
        // pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
        // "over" operator for front-to-back blending
		sum = sum + col;//*(1.0f - sum.w);

        t += tstep;
        if (t > tfar) break;
		
        pos += step;
    }

    sum *= brightness;

    d_iColors[y*imageW + x] = sum;
 
    d_iRed[y*imageW + x] = sum.x;
    d_iGreen[y*imageW + x] = sum.y;
    d_iBlue[y*imageW + x] = sum.z;
}

__global__
void create_image(uint *output, float4 *d_iColors, float global_max, float red, float green, float blue, uint imageW, uint imageH){

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((x >= imageW) || (y >= imageH)) return;

    output[y*imageH+x] = rgbaFloatToInt(d_iColors[y*imageW+x], global_max, red, green, blue);
} 

void setup_cluster(void *cluster, hipExtent volumeSize, uint image_size, hipArray *d_volumeArray_cluster){
	
	// Cluster setup

		// create 3D array
		hipChannelFormatDesc channelDesc_cluster = hipCreateChannelDesc<VolumeType>();
		cutilSafeCall( hipMalloc3DArray(&d_volumeArray_cluster, &channelDesc_cluster, volumeSize) );

		// copy data to 3D array
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr(cluster, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
		copyParams.dstArray = d_volumeArray_cluster;
		copyParams.extent   = volumeSize;
		copyParams.kind     = hipMemcpyHostToDevice;
		cutilSafeCall( hipMemcpy3D(&copyParams) );  

		// set texture parameters
		tex_cluster.normalized = true;                      // access with normalized texture coordinates
		tex_cluster.filterMode = hipFilterModePoint;      // linear interpolation
		tex_cluster.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
		tex_cluster.addressMode[1] = hipAddressModeClamp;

		// bind array to 3D texture
		cutilSafeCall(hipBindTextureToArray(tex_cluster, d_volumeArray_cluster, channelDesc_cluster));
	
}

void setup_volume(void *h_volume, hipExtent volumeSize, uint image_size, hipArray *d_volumeArray){
	
	// create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    cutilSafeCall( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );  

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModePoint;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint *d_cluster, float* d_iRed, float* d_oRed, 
						float* d_iGreen, float* d_oGreen, float* d_iBlue, float* d_oBlue, float4* d_iColors, unsigned short* data, 
						unsigned short *cluster_data, uint imageW, uint imageH, float density, float brightness, 
						float4 one, float4 two, float4 three, float4 four, float4 five, float4 six,
						void *h_volume, void *cluster, hipExtent volumeSize, hipArray *d_volumeArray, hipArray *d_volumeArray_cluster, int *set)
{	
		
	int size = imageH * imageW;
	
	if(set[0] == 0){
		setup_volume(h_volume, volumeSize, size, d_volumeArray);
		set[0] = 1;
	}
	if(set[1] == 0){
		setup_cluster(cluster, volumeSize, size, d_volumeArray_cluster);
		set[1] = 1;
	}
	/* clear colors buffers */

	cutilSafeCall(hipMemset(d_iColors, 0, imageH*imageW*sizeof(float4)));	
	
	cutilSafeCall(hipMemset(d_iRed, 0, imageH*imageW*sizeof(float)));	
	cutilSafeCall(hipMemset(d_oRed, 0, imageH*imageW*sizeof(float)));	
	cutilSafeCall(hipMemset(d_iGreen, 0, imageH*imageW*sizeof(float)));	
	cutilSafeCall(hipMemset(d_oGreen, 0, imageH*imageW*sizeof(float)));
	cutilSafeCall(hipMemset(d_iBlue, 0, imageH*imageW*sizeof(float)));	
	cutilSafeCall(hipMemset(d_oBlue, 0, imageH*imageW*sizeof(float)));

	d_render<<<gridSize, blockSize>>>(d_iColors, data, d_iRed, d_iGreen, d_iBlue, imageW, imageH, density, brightness, 
						one, two, three, four, five, six, 0);

	float max_red = reduce_max(d_oRed, d_iRed, size);
	float max_green = reduce_max(d_oGreen, d_iGreen, size);
	float max_blue = reduce_max(d_oBlue, d_iBlue, size);
	
	float global_max = fmax(max_red, max_green);
	global_max = fmax(global_max, max_blue);
	
	create_image<<<gridSize, blockSize>>>(d_output, d_iColors, global_max, max_red, max_green, max_blue, imageW, imageH);
	
	// render image
	// 
	d_render<<<gridSize, blockSize>>>(d_iColors, cluster_data, d_iRed, d_iGreen, d_iBlue, imageW, imageH, density, brightness, 
					one, two, three, four, five, six, 1);

	max_red = reduce_max(d_oRed, d_iRed, size);
	max_green = reduce_max(d_oGreen, d_iGreen, size);
	max_blue = reduce_max(d_oBlue, d_iBlue, size);

	global_max = fmax(max_red, max_green);
	global_max = fmax(global_max, max_blue);

	create_image<<<gridSize, blockSize>>>(d_cluster, d_iColors, global_max, max_red, max_green, max_blue, imageW, imageH);	  
}
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix) );
}

#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
